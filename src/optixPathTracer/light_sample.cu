#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu_matrix_namespace.h>
#include "helpers.h"
#include "prd.h"
#include "random.h"
#include "rt_function.h"
#include "material_parameters.h"
#include "light_parameters.h"
#include "state.h"
#include <assert.h>
#include <stdio.h>

using namespace optix;

rtDeclareVariable(int, sysNumberOfLights, , );

rtBuffer<LightParameter> sysLightParameters;

RT_FUNCTION float3 UniformSampleSphere(float u1, float u2)
{
	float z = 1.f - 2.f * u1;
	float r = sqrtf(max(0.f, 1.f - z * z));
	float phi = 2.f * M_PIf * u2;
	float x = r * cosf(phi);
	float y = r * sinf(phi);

	return make_float3(x, y, z);
}


RT_CALLABLE_PROGRAM void envmap_sample(const LightParameter &light, const float3 &surfacePos, unsigned int &seed, LightSample &lightSample)
{
	const float r1 = rnd(seed);
	const float r2 = rnd(seed);

	const unsigned int sizeU = static_cast<unsigned int>(light.idEnvironmentCDF_U.size().x);
	const unsigned int sizeV = static_cast<unsigned int>(light.idEnvironmentCDF_V.size());

	unsigned int ilo = 0; // lower limit
	unsigned int ihi = sizeV - 1; // higher limit

	//printf("(GPU) type: %d", light.lightType);
	//printf("(GPU) textureID: %d\n", light.idEnvironmentTexture);
	//printf("(GPU) Envmap integral: %f\n", sysLightParameters[0].environmentIntegral);

	while (ilo != ihi - 1)
	{
		const unsigned int i = (ilo + ihi) >> 1;
		//printf("%u %u\n", light.idEnvironmentCDF_U.size().x, light.idEnvironmentCDF_U.size().y);
		const float cdf = light.idEnvironmentCDF_V[i];
		//assert(LightType::ENVMAP != light.lightType);
		if (r2 < cdf) // If the cdf is greater than the sample, use that as new higher limit.
		{
			ihi = i;
		}
		else // If the sample is greater than or equal to the CDF value, use that as new lower limit.
		{
			ilo = i;
		}
	}
	
	uint2 index;
	index.y = ilo;

	ilo = 0;
	ihi = sizeU - 1;

	while (ilo != ihi - 1)
	{
		index.x = (ilo + ihi) >> 1;
		const float cdf = light.idEnvironmentCDF_U[index];
		if (r1 < cdf) // If the cdf is greater than the sample, use that as new higher limit.
		{
			ihi = index.x;
		}
		else // If the sample is greater than or equal to the CDF value, use that as new lower limit.
		{
			ilo = index.x;
		}
	}
	
	index.x = ilo;
	
	// Continuous sampling of the CDF.
	// Continuous sampling of the CDF.
	const float cdfLowerU = light.idEnvironmentCDF_U[index];
	const float cdfUpperU = light.idEnvironmentCDF_U[make_uint2(index.x + 1, index.y)];
	const float du = (r1- cdfLowerU) / (cdfUpperU - cdfLowerU);

	const float cdfLowerV = light.idEnvironmentCDF_V[index.y];
	const float cdfUpperV = light.idEnvironmentCDF_V[index.y + 1];
	const float dv = (r2 - cdfLowerV) / (cdfUpperV - cdfLowerV);

	// Texture lookup coordinates.
	const float u = (float(index.x) + du) / float(sizeU - 1);
	const float v = (float(index.y) + dv) / float(sizeV - 1);

	// Light sample direction vector polar coordinates. This is where the environment rotation happens!
	// DAR FIXME Use a light.matrix to rotate the resulting vector instead.
	const float phi = (u /* - 0.5f */) * 2.0f * M_PIf;
	const float theta = v * M_PIf; // theta == 0.0f is south pole, theta == M_PIf is north pole.

	const float sinTheta = sinf(theta);
	// The miss program places the 1->0 seam at the positive z-axis and looks from the inside.
	lightSample.direction = make_float3(-sinf(phi) * sinTheta,  // Starting on positive z-axis going around clockwise (to negative x-axis).
		-cosf(theta),           // From south pole to north pole.
		cosf(phi) * sinTheta); // Starting on positive z-axis.

	// Note that environment lights do not set the light sample position!
	lightSample.distance = RT_DEFAULT_MAX; // Environment light.

	const float3 emission = make_float3(optix::rtTex2D<float4>(light.idEnvironmentTexture, u, v));
	// Explicit light sample. The returned emission must be scaled by the inverse probability to select this light.
	lightSample.emission = emission * float(sysNumberOfLights);
	// For simplicity we pretend that we perfectly importance-sampled the actual texture-filtered environment map
	// and not the Gaussian-smoothed one used to actually generate the CDFs and uniform sampling in the texel.
	lightSample.pdf = 0.3333333333f * (emission.x + emission.y + emission.z) / light.environmentIntegral;
}


RT_CALLABLE_PROGRAM void sphere_sample(const LightParameter &light, const float3 &surfacePos, unsigned int &seed, LightSample &lightSample)
{
	const float r1 = rnd(seed);
	const float r2 = rnd(seed);
	
	lightSample.pdf = 0.0f;

	optix::float3 lightSamplePos = light.position + UniformSampleSphere(r1, r2) * light.radius;
	lightSample.direction = lightSamplePos - surfacePos;
	lightSample.distance = length(lightSample.direction);

	if (1.0e-6f < lightSample.distance)
	{
		lightSample.direction /= lightSample.distance;
		optix::float3 lightNormal = normalize(lightSamplePos - light.position);

		const float cosTheta = dot(-lightSample.direction, lightNormal); // light�� surface�� �ٶ󺸴� �����̾�� �Ѵ�.
		if (1.0e-6f < cosTheta)
		{
			lightSample.emission = light.emission * float(sysNumberOfLights);
			lightSample.pdf = (lightSample.distance * lightSample.distance) / (light.area * cosTheta);
		}
	}
}


/*
 lightSample�� pdf, distance, direction, emission�� ���Ѵ�.
 pdf: Monte Carlo estimator�� power heuristic ����� ����ϱ� ����
 distance: ���� ���� �� �ִ� valid�� �������� Ȯ���ϱ� ���� shadowRay�� �� ����
 direction: shadowRay�� �� �Ÿ�
 emission: emissive radiance
 */
RT_CALLABLE_PROGRAM void quad_sample(const LightParameter &light, const float3 &surfacePos, unsigned int &seed, LightSample &lightSample)
{
	const float r1 = rnd(seed);
	const float r2 = rnd(seed);

	lightSample.pdf = 0.0f;

	optix::float3 lightSamplePos = light.position + light.u * r1 + light.v * r2;
	lightSample.direction = lightSamplePos - surfacePos;
	lightSample.distance = length(lightSample.direction);

	if (1.0e-6f < lightSample.distance)
	{
		lightSample.direction /= lightSample.distance;

		const float cosTheta = dot(-lightSample.direction, light.normal); // light�� surface�� �ٶ󺸴� �����̾�� �Ѵ�.
		if (1.0e-6f < cosTheta)
		{
			lightSample.emission = light.emission * float(sysNumberOfLights);
			lightSample.pdf = (lightSample.distance * lightSample.distance) / (light.area * cosTheta);
		}
	}
}
