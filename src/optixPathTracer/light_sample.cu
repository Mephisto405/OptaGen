#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu_matrix_namespace.h>
#include "helpers.h"
#include "prd.h"
#include "random.h"
#include "rt_function.h"
#include "material_parameters.h"
#include "light_parameters.h"
#include "state.h"

using namespace optix;

rtDeclareVariable(int, sysNumberOfLights, , );

RT_FUNCTION float3 UniformSampleSphere(float u1, float u2)
{
	float z = 1.f - 2.f * u1;
	float r = sqrtf(max(0.f, 1.f - z * z));
	float phi = 2.f * M_PIf * u2;
	float x = r * cosf(phi);
	float y = r * sinf(phi);

	return make_float3(x, y, z);
}

RT_CALLABLE_PROGRAM void sphere_sample(LightParameter &light, PerRayData_radiance &prd, LightSample &sample)
{
	const float r1 = rnd(prd.seed);
	const float r2 = rnd(prd.seed);
	sample.surfacePos = light.position + UniformSampleSphere(r1, r2) * light.radius;
	sample.normal = normalize(sample.surfacePos - light.position);
	sample.emission = light.emission * sysNumberOfLights;
}

RT_CALLABLE_PROGRAM void quad_sample(LightParameter &light, PerRayData_radiance &prd, LightSample &sample)
{
	const float r1 = rnd(prd.seed);
	const float r2 = rnd(prd.seed);
	sample.surfacePos = light.position + light.u * r1 + light.v * r2;
	sample.normal = light.normal;
	sample.emission = light.emission * sysNumberOfLights;
}
