#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu_matrix_namespace.h>
#include "helpers.h"
#include "prd.h"
#include "random.h"
#include "rt_function.h"
#include "material_parameters.h"
#include "state.h"

using namespace optix;
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

// -----------------------------------------------------------------------------

RT_FUNCTION float fresnel( float cos_theta_i, float cos_theta_t, float eta )
{
    const float rs = ( cos_theta_i - cos_theta_t*eta ) / 
                     ( cos_theta_i + eta*cos_theta_t );
    const float rp = ( cos_theta_i*eta - cos_theta_t ) /
                     ( cos_theta_i*eta + cos_theta_t );

    return 0.5f * ( rs*rs + rp*rp );
}


RT_FUNCTION float3 logf( float3 v )
{
    return make_float3( logf(v.x), logf(v.y), logf(v.z) );
}


RT_CALLABLE_PROGRAM void Pdf(MaterialParameter &mat, State &state, PerRayData_radiance &prd)
{
	prd.pdf = 1.0f;
}


RT_CALLABLE_PROGRAM void Sample(MaterialParameter &mat, State &state, PerRayData_radiance &prd)
{
	const float3 w_out = prd.wo;
	float3 normal = state.normal;
	float cos_theta_i = optix::dot( w_out, normal );
	const float IOR = mat.intIOR / mat.extIOR;

	float eta;
	float3 transmittance = make_float3(1.0f);
	float3 extinction = -logf(make_float3(0.905f, 0.63f, 0.3));
	if( cos_theta_i > 0.0f )
	{
		eta = IOR;
	} 
	else
	{
		transmittance = optix::expf(-extinction * t_hit);
		eta = 1.0f / IOR;
		cos_theta_i = -cos_theta_i;
		normal = -normal;
	}
	//intData.mat.color = transmittance;

	float3 w_t;
	const bool tir  = !optix::refract( w_t, -w_out, normal, eta );
	const float cos_theta_t = -optix::dot( normal, w_t );
	const float R  = tir  ? 1.0f : fresnel( cos_theta_i, cos_theta_t, eta );

	const float z = rnd(prd.seed);
	if( z <= R )
	{
		// Reflect
		prd.origin = state.fhp;
		prd.bsdfDir =  optix::reflect( -w_out, normal );

		// update path feature
		prd.roughness = 0.0f;
		prd.tag = REFL;
	}
	else
	{
		// Refract
		prd.origin = state.bhp;
		prd.bsdfDir = w_t;

		// update path feature
		prd.roughness = 0.0f;
		prd.tag = TRAN;
	}
}


RT_CALLABLE_PROGRAM float3 Eval(MaterialParameter &mat, State &state, PerRayData_radiance &prd)
{
	/* World frame vectors */
	const float3 N = state.normal;
	const float3 i = prd.wo;
	const float3 o = prd.bsdfDir;

	/* Roughness scaling and conversion */
	const float iDotN = dot(i, N);

	if (iDotN * dot(o, N) > 0) // reflection
	{
		// update path feature
		prd.thpt_at_vtx = make_float3(1.0f);

		return prd.thpt_at_vtx;
	}
	else
	{
		// update path feature
		prd.thpt_at_vtx = mat.color;

		return prd.thpt_at_vtx;
	}
}

