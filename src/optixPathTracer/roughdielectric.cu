#include "hip/hip_runtime.h"
/*
Rough dielectric materials
https://dl.acm.org/citation.cfm?id=2383874 [Walter et al.]
*/

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu_matrix_namespace.h>
#include "helpers.h"
#include "prd.h"
#include "random.h"
#include "rt_function.h"
#include "material_parameters.h"
#include "state.h"

using namespace optix;

/* HELPERS */

/* Return 1 if the input >= 0 and -1 otherwise. */
RT_FUNCTION inline float sgn(float x)
{
	return (x >= 0.0f) ? 1.0f : -1.0f;
}

/* Convert the user-specified roughness to the roughness value suitable
   for each model. This is done in a way such that even different models
   produce similar appearances for the same user-specified roughness. */
RT_FUNCTION float alphaConversion(float roughness, DistType dist)
{
	const float minAlpha = 1e-3f;
	const float b2g = 1.1312f; // Walter et al. (Fig. 12)
	float ggx_alpha = fmax(roughness, minAlpha);
	float beck_alpha = ggx_alpha / b2g;

	switch (dist)
	{
	case DistType::Beckmann:
		return beck_alpha;
	case DistType::GGX:
		return ggx_alpha;
	case DistType::Phong:
		return 2.0f / (beck_alpha * beck_alpha) - 2.0f;
	default:
		return ggx_alpha;
	}
}

/* Sample the polar angle and the azimuthal angle.
   Then convert them into the 3D Euclidean coordinates (local frame). */
RT_FUNCTION float3 sample(float r1, float r2, float alpha, DistType dist)
{
	float cosTheta = 0.0f; // cosine of the polar angle
	const float phi = r1 * 2.0f * M_PIf; // azimuthal angle
	const float alphaSqr = alpha * alpha;

	switch (dist)
	{
	case DistType::Beckmann:
	{
		cosTheta = 1.0f / sqrtf(fmax(1.0f - alphaSqr * logf(1.0f - r2), 0.0f));
		break;
	}
	case DistType::GGX:
	{
		cosTheta = 1.0f / sqrtf(fmax(1.0f + alphaSqr * r2 / (1.0f - r2), 0.0f));
		break;
	}
	case DistType::Phong:
	{
		cosTheta = pow(r2, 1.0f / (alpha + 2.0f));
		break;
	}
	}

	const float r = sqrtf(fmax(1.0f - cosTheta * cosTheta, 0.0f));
	return optix::make_float3(cosf(phi) * r, sinf(phi) * r, cosTheta);
}

/* Fresnel (reflection) coefficient.
   This term describes how much of an electromagnetic wave is reflected
   by an impedance discontinuity in the transmission medium
   [Wikipedia: Reflection coefficient]. */
RT_FUNCTION float fresnelTerm(float cosThetaI, float invEta, float &cosThetaT)
{
	if (cosThetaI < 0.0f)
		cosThetaI = -cosThetaI;
	const float cosThetaTSqr = 1.0f - invEta * invEta * (1.0f - cosThetaI * cosThetaI);
	if (cosThetaTSqr < 0.0f) // total reflection
	{
		cosThetaT = 0.0f; // meaningless
		return 1.0f;
	}

	cosThetaT = sqrtf(fmax(cosThetaTSqr, 0.0f)); // = abs(cosThetaT), to be precise
	const float Rs = (cosThetaT - invEta * cosThetaI) / (cosThetaT + invEta * cosThetaI);
	const float Rp = (invEta * cosThetaT - cosThetaI) / (invEta * cosThetaT + cosThetaI);
	return 0.5f * (Rs * Rs + Rp * Rp);
}

/* Wrapper for the Fresnel coefficient. */
RT_FUNCTION float fresnelTerm(float cosThetaI, float invEta)
{
	float cosThetaT;
	return fresnelTerm(cosThetaI, invEta, cosThetaT);
}

/* Microfacet distribution function. */
RT_FUNCTION float D(float cosThetaM, float alpha, DistType dist)
{
	if (cosThetaM <= 0.0f)
		return 0.0f;

	const float alphaSqr = alpha * alpha;
	const float cosThetaSqr = cosThetaM * cosThetaM;
	const float cosThetaQd = cosThetaSqr * cosThetaSqr;
	const float beckmannExp = -(1.0f / cosThetaSqr - 1.0f) / alphaSqr;
	const float ggxDivisor = (1.0f - beckmannExp);

	switch (dist)
	{
	case DistType::Beckmann:
		return optix::expf(beckmannExp) * M_1_PIf / (alphaSqr * cosThetaQd);
	case DistType::GGX:
		return M_1_PIf / (cosThetaQd * ggxDivisor * ggxDivisor);
	case DistType::Phong:
		return 0.5f * (alpha + 2) * M_1_PIf * powf(cosThetaM, alpha);
	default:
		return 0.0f;
	}
}

/* Unidirectional shadow-masking function. */
RT_FUNCTION float G1(float3 v, float3 m, float3 n, float alpha, DistType dist)
{
	const float cosTheta = dot(v, n);
	if (dot(v, m) / cosTheta <= 0.0f)
		return 0.0f;

	const float tanTheta = abs(sqrtf(fmax(1 - cosTheta * cosTheta, 0.0f)) / cosTheta); // = abs(invTanTheta), to be precise
	const float alphaTan = alpha * tanTheta;
	float a;
	if (dist == DistType::Beckmann)
		a = 1.0f / alphaTan;
	else if (dist == DistType::Phong)
		a = sqrtf(1 + 0.5f * alpha) / tanTheta;

	switch (dist)
	{
	case DistType::Beckmann:
		if (a < 1.6f)
			return (3.535f * a + 2.181f * a * a) / (1.0f + 2.276f * a + 2.577f * a * a);
		else
			return 1.0f;
	case DistType::GGX:
		return 2.0f / (1.0f + sqrtf(1.0f + alphaTan * alphaTan));
	case DistType::Phong:
		if (a < 1.6f)
			return (3.535f * a + 2.181f * a * a) / (1.0f + 2.276f * a + 2.577f * a * a);
		else
			return 1.0f;
	default:
		return 0.0f;
	}
}

/* Bidirectional shadow-masking function. */
RT_FUNCTION float G(float3 i, float3 o, float3 m, float3 n, float alpha, DistType dist)
{
	const float g1 = G1(i, m, n, alpha, dist);
	if (g1 == 0.0f)
		return 0.0f;

	const float g2 = G1(o, m, n, alpha, dist);
	if (g2 == 0.0f)
		return 0.0f;

	return abs(g1 * g2); // take abs to prevent unintentional numerical errors
}


/* MAIN ROUTINES */

/* Sample an transmitted or an reflected directional vector. */
RT_CALLABLE_PROGRAM void Sample(MaterialParameter &mat, State &state, PerRayData_radiance &prd)
{
	/* World frame vectors */
	const float3 N = state.normal; // shading normal, pointing to the outside
	const float3 V = prd.wo; // = -ray.direction, pointing away from the intersection point

	/* Roughness scaling and conversion */
	const float VDotN = optix::dot(V, N);
	float sampleAlphaScale = 1.2f - 0.2f * sqrtf(abs(VDotN)); // roughness scaling trick by Walter et al. (Chap. 5.3, p.8)
	float sampleAlpha = alphaConversion(sampleAlphaScale * mat.roughness, mat.dist);

	/* Microfacet-normal sampling */
	const float r1 = rnd(prd.seed);
	const float r2 = rnd(prd.seed);
	float3 m = sample(r1, r2, sampleAlpha, mat.dist); // local frame
	optix::Onb onb(N);
	onb.inverse_transform(m); // covnert a local frame to the world frame

	/* Fresnel term computing */
	const float VDotM = optix::dot(V, m);
	float cosThetaT = 0.0f; // transmission angle ([0, pi/2])
	const float invEta = VDotM > 0.0f ? mat.extIOR / mat.intIOR : mat.intIOR / mat.extIOR;
	const float F = fresnelTerm(VDotM, invEta, cosThetaT);

	const float p = rnd(prd.seed);
	if (p <= F)
	{
		prd.origin = state.fhp;
		prd.bsdfDir = 2.0f * VDotM * m - V;

		// update path feature
		prd.roughness = alphaConversion(mat.roughness, mat.dist);
		prd.bounce_type = BSDF_REFLECTION | BSDF_GLOSSY;

		/* Sanity check */
		if (dot(V, N) * dot(prd.bsdfDir, N) <= 0.0f) // should be reflected, but it wasn't
			prd.done = true;
	}
	else
	{
		prd.origin = state.bhp;
		prd.bsdfDir = (invEta * VDotM - sgn(VDotM) * cosThetaT) * m - invEta * V;

		// update path feature
		prd.roughness = alphaConversion(mat.roughness, mat.dist);
		prd.bounce_type = BSDF_TRANSMISSION | BSDF_GLOSSY;

		/* Sanity check */
		if (dot(V, N) * dot(prd.bsdfDir, N) >= 0.0f) // should be refracted, but it wasn't
			prd.done = true;
	}
}

/* Evaluate pdf (sampled direction). */
RT_CALLABLE_PROGRAM void Pdf(MaterialParameter &mat, State &state, PerRayData_radiance &prd)
{
	/* Sanity check */
	if (prd.done)
	{
		prd.pdf = 0.0f;
		return;
	}

	/* World frame vectors */
	const float3 N = state.normal;
	const float3 i = prd.wo;
	const float3 o = prd.bsdfDir;

	/* Roughness scaling and conversion */
	const float iDotN = dot(i, N);
	float sampleAlphaScale = 1.2f - 0.2f * sqrtf(abs(iDotN)); // roughness scaling trick by Walter et al. (Chap. 5.3, p.8)
	float sampleAlpha = alphaConversion(sampleAlphaScale * mat.roughness, mat.dist);

	if (iDotN * dot(o, N) > 0) // reflection
	{
		/* Half vector */
		const float3 m = normalize(i + o) * sgn(iDotN);

		/* Fresnel term computing */
		const float iDotm = dot(i, m);
		const float invEta = iDotm > 0.0f ? mat.extIOR / mat.intIOR : mat.intIOR / mat.extIOR;
		const float F = fresnelTerm(iDotm, invEta);

		/* Microfacet distribution evaluating */
		const float mDotN = dot(m, N);
		const float microPdf = D(mDotN, sampleAlpha, mat.dist);

		/* Macrosurface distribution evaluating */
		prd.pdf = abs(F * microPdf * mDotN / (4 * iDotm));
	}
	else // refraction
	{
		/* Half vector */
		const float eta = iDotN > 0.0f ? mat.intIOR / mat.extIOR : mat.extIOR / mat.intIOR;
		const float3 m = -normalize(i + eta * o);

		/* Fresnel term computing */
		const float iDotm = dot(i, m);
		const float invEta = iDotm > 0.0f ? mat.extIOR / mat.intIOR : mat.intIOR / mat.extIOR;
		const float F = fresnelTerm(iDotm, invEta);

		/* Microfacet distribution evaluating */
		const float mDotN = dot(m, N);
		const float microPdf = D(mDotN, sampleAlpha, mat.dist);

		/* Macrosurface distribution evaluating */
		const float oDotm = dot(o, m);
		const float divisor = iDotm / eta + oDotm;
		prd.pdf = abs((1 - F) * microPdf * mDotN * oDotm / (divisor * divisor));
	}
}

/* Evaluate f_s(i,o,n)*|o*n|. */
RT_CALLABLE_PROGRAM float3 Eval(MaterialParameter &mat, State &state, PerRayData_radiance &prd)
{
	/* Sanity check */
	if (prd.done)
	{
		return make_float3(0.0f);
	}

	/* World frame vectors */
	const float3 N = state.normal;
	const float3 i = prd.wo;
	const float3 o = prd.bsdfDir;

	/* Roughness scaling and conversion */
	const float iDotN = dot(i, N);
	float alpha = alphaConversion(mat.roughness, mat.dist);

	if (iDotN * dot(o, N) > 0) // reflection
	{
		/* Half vector */
		const float3 m = normalize(i + o) * sgn(iDotN);

		/* Fresnel term computing */
		const float iDotm = dot(i, m);
		const float invEta = iDotm > 0.0f ? mat.extIOR / mat.intIOR : mat.intIOR / mat.extIOR;
		const float F = fresnelTerm(iDotm, invEta);

		/* Microfacet distribution evaluating */
		const float mDotN = dot(m, N);
		const float microPdf = D(mDotN, alpha, mat.dist);

		/* Bidirectional shadow-masking function */
		const float Geo = G(i, o, m, N, alpha, mat.dist);

		/* BSDF*cosine evaluating */
		const float f = abs(F * Geo * microPdf / (4 * iDotN));

		// update path feature
		prd.thpt_at_vtx = make_float3(f);

		return prd.thpt_at_vtx;
	}
	else // refraction
	{
		/* Half vector */
		const float eta = iDotN > 0.0f ? mat.intIOR / mat.extIOR : mat.extIOR / mat.intIOR;
		const float3 m = -normalize(i + eta * o);

		/* Fresnel term computing */
		const float iDotm = dot(i, m);
		const float invEta = iDotm > 0.0f ? mat.extIOR / mat.intIOR : mat.intIOR / mat.extIOR;
		const float F = fresnelTerm(iDotm, invEta);

		/* Microfacet distribution evaluating */
		const float mDotN = dot(m, N);
		const float microPdf = D(mDotN, alpha, mat.dist);

		/* Bi-directional shadow-masking function */
		const float Geo = G(i, o, m, N, alpha, mat.dist);

		/* BSDF*cosine evaluating */
		const float oDotm = dot(o, m);
		const float divisor = iDotm / eta + oDotm;
		const float f = abs((1 - F) * Geo * microPdf * iDotm * oDotm / (divisor * divisor) / iDotN);

		// update path feature
		prd.thpt_at_vtx = mat.color * make_float3(f);

		return prd.thpt_at_vtx;
	}
}


RT_CALLABLE_PROGRAM float3 EvalDiffuse(MaterialParameter &mat, State &state, PerRayData_radiance &prd)
{
	return make_float3(0.0f);
}