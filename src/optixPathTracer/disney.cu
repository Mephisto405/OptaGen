#include "hip/hip_runtime.h"
/*
 Copyright Disney Enterprises, Inc.  All rights reserved.

 Licensed under the Apache License, Version 2.0 (the "License");
 you may not use this file except in compliance with the License
 and the following modification to it: Section 6 Trademarks.
 deleted and replaced with:

 6. Trademarks. This License does not grant permission to use the
 trade names, trademarks, service marks, or product names of the
 Licensor and its affiliates, except as required for reproducing
 the content of the NOTICE file.

 You may obtain a copy of the License at
 http://www.apache.org/licenses/LICENSE-2.0
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu_matrix_namespace.h>
#include "helpers.h"
#include "prd.h"
#include "random.h"
#include "rt_function.h"
#include "material_parameters.h"
#include "state.h"

using namespace optix;

RT_FUNCTION float sqr(float x) { return x*x; }

RT_FUNCTION float SchlickFresnel(float u)
{
	float m = clamp(1.0f - u, 0.0f, 1.0f);
	float m2 = m*m;
	return m2*m2*m; // pow(m,5)
}

/* clearcoat lobe */
RT_FUNCTION float GTR1(float NDotH, float a)
{
	if (a >= 1.0f) return (1.0f / M_PIf);
	float a2 = a*a;
	float t = 1.0f + (a2 - 1.0f)*NDotH*NDotH;
	return (a2 - 1.0f) / (M_PIf*logf(a2)*t);
}

/* specular lobe */
RT_FUNCTION float GTR2(float NDotH, float a)
{
	float a2 = a*a;
	float t = 1.0f + (a2 - 1.0f)*NDotH*NDotH;
	return a2 / (M_PIf * t*t);
}

RT_FUNCTION float smithG_GGX(float NDotv, float alphaG)
{
	float a = alphaG*alphaG;
	float b = NDotv*NDotv;
	return 1.0f / (NDotv + sqrtf(a + b - a*b));
}


/*
	https://disney-animation.s3.amazonaws.com/library/s2012_pbs_disney_brdf_notes_v2.pdf
	http://simon-kallweit.me/rendercompo2015/report/
	*/
RT_CALLABLE_PROGRAM void Pdf(MaterialParameter &mat, State &state, PerRayData_radiance &prd)
{
	float3 n = state.ffnormal;
	float3 V = prd.wo;
	float3 L = prd.bsdfDir;

	float specularAlpha = max(0.001f, mat.roughness);
	float clearcoatAlpha = lerp(0.1f, 0.001f, mat.clearcoatGloss);

	float diffuseRatio = 0.5f * (1.f - mat.metallic);
	float specularRatio = 1.f - diffuseRatio;

	float3 half = normalize(L + V);

	float cosTheta = abs(dot(half, n));
	float pdfGTR2 = GTR2(cosTheta, specularAlpha) * cosTheta;
	float pdfGTR1 = GTR1(cosTheta, clearcoatAlpha) * cosTheta;

	// calculate diffuse and specular pdfs and mix ratio
	float ratio = 1.0f / (1.0f + mat.clearcoat);
	float pdfSpec = lerp(pdfGTR1, pdfGTR2, ratio) / (4.0 * abs(dot(L, half)));
	float pdfDiff = abs(dot(L, n))* (1.0f / M_PIf);

	// weight pdfs according to ratios
	prd.pdf = diffuseRatio * pdfDiff + specularRatio * pdfSpec;

}


/*
	https://disney-animation.s3.amazonaws.com/library/s2012_pbs_disney_brdf_notes_v2.pdf
	http://simon-kallweit.me/rendercompo2015/report/
	https://learnopengl.com/PBR/IBL/Specular-IBL
	*/
RT_CALLABLE_PROGRAM void Sample(MaterialParameter &mat, State &state, PerRayData_radiance &prd)
{
	float3 N = state.ffnormal;
	float3 V = prd.wo;
	prd.origin = state.fhp;

	float3 dir;

	float probability = rnd(prd.seed);
	float diffuseRatio = 0.5f * (1.0f - mat.metallic);

	float r1 = rnd(prd.seed);
	float r2 = rnd(prd.seed);

	optix::Onb onb(N); // basis

	float a = max(0.001f, mat.roughness);

	if (probability < diffuseRatio) // sample diffuse
	{
		cosine_sample_hemisphere(r1, r2, dir);
		onb.inverse_transform(dir);

		// update path feature
		prd.roughness = a;
		prd.bounce_type = BSDF_REFLECTION | BSDF_DIFFUSE;
	}
	else
	{
		// GTR2 sampling
		float phi = r1 * 2.0f * M_PIf;

		float cosTheta = sqrtf((1.0f - r2) / (1.0f + (a*a - 1.0f) *r2)); // GGX sampling (roughdielectric.cu)
		float sinTheta = sqrtf(1.0f - (cosTheta * cosTheta));
		float sinPhi = sinf(phi);
		float cosPhi = cosf(phi);

		float3 half = make_float3(sinTheta*cosPhi, sinTheta*sinPhi, cosTheta);
		onb.inverse_transform(half);

		dir = 2.0f*dot(V, half)*half - V; //reflection vector

		// update path feature
		prd.roughness = a;
		prd.bounce_type = BSDF_REFLECTION | BSDF_GLOSSY;
	}
	prd.bsdfDir = dir;
}


/*
	https://disney-animation.s3.amazonaws.com/library/s2012_pbs_disney_brdf_notes_v2.pdf
	http://simon-kallweit.me/rendercompo2015/report/
	https://github.com/wdas/brdf/blob/master/src/brdfs/disney.brdf
	*/
RT_CALLABLE_PROGRAM float3 Eval(MaterialParameter &mat, State &state, PerRayData_radiance &prd)
{
	float3 N = state.ffnormal;
	float3 V = prd.wo;
	float3 L = prd.bsdfDir;

	float NDotL = dot(N, L);
	float NDotV = dot(N, V);
	if (NDotL <= 0.0f || NDotV <= 0.0f) return make_float3(0.0f);

	float3 H = normalize(L + V);
	float NDotH = dot(N, H);
	float LDotH = dot(L, H);

	float3 Cdlin = mat.color;
	float Cdlum = 0.3f*Cdlin.x + 0.6f*Cdlin.y + 0.1f*Cdlin.z; // luminance approx.

	float3 Ctint = Cdlum > 0.0f ? Cdlin / Cdlum : make_float3(1.0f); // normalize lum. to isolate hue+sat
	float3 Cspec0 = lerp(mat.specular*0.08f*lerp(make_float3(1.0f), Ctint, mat.specularTint), Cdlin, mat.metallic);
	float3 Csheen = lerp(make_float3(1.0f), Ctint, mat.sheenTint);

	// Diffuse fresnel - go from 1 at normal incidence to .5 at grazing
	// and mix in diffuse retro-reflection based on roughness
	float FL = SchlickFresnel(NDotL), FV = SchlickFresnel(NDotV);
	float Fd90 = 0.5f + 2.0f * LDotH * LDotH * mat.roughness;
	float Fd = lerp(1.0f, Fd90, FL) * lerp(1.0f, Fd90, FV);

	// Based on Hanrahan-Krueger brdf approximation of isotropic bssrdf
	// 1.25 scale is used to (roughly) preserve albedo
	// Fss90 used to "flatten" retroreflection based on roughness
	float Fss90 = LDotH*LDotH*mat.roughness;
	float Fss = lerp(1.0f, Fss90, FL) * lerp(1.0f, Fss90, FV);
	float ss = 1.25f * (Fss * (1.0f / (NDotL + NDotV) - 0.5f) + 0.5f);

	// specular 
	float a = max(0.001f, mat.roughness); // Section 5.4 of the first ref.
	float Ds = GTR2(NDotH, a);
	float FH = SchlickFresnel(LDotH);
	float3 Fs = lerp(Cspec0, make_float3(1.0f), FH);
	float roughg = sqr(mat.roughness*0.5f + 0.5f); // Section 5.6 of the first ref.
	float Gs = smithG_GGX(NDotL, roughg) * smithG_GGX(NDotV, roughg);

	// sheen
	float3 Fsheen = FH * mat.sheen * Csheen;

	// clearcoat (ior = 1.5 -> F0 = 0.04)
	float Dr = GTR1(NDotH, lerp(0.1f, 0.001f, mat.clearcoatGloss));
	float Fr = lerp(0.04f, 1.0f, FH);
	float Gr = smithG_GGX(NDotL, 0.25f) * smithG_GGX(NDotV, 0.25f); // Section 5.6 of the first ref.

	float3 out = ((1.0f / M_PIf) * lerp(Fd, ss, mat.subsurface)*Cdlin + Fsheen)
		* (1.0f - mat.metallic)
		+ Gs*Fs*Ds + 0.25f*mat.clearcoat*Gr*Fr*Dr;

	// update path feature
	prd.thpt_at_vtx = out * clamp(dot(N, L), 0.0f, 1.0f);

	return prd.thpt_at_vtx;
}