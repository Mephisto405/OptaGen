#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu_matrix_namespace.h>
#include "helpers.h"
#include "prd.h"
#include "random.h"
#include "rt_function.h"
#include "material_parameters.h"
#include "light_parameters.h"
#include "state.h"
#include <assert.h>

using namespace optix;

rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable(float3, back_hit_point, attribute back_hit_point, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(int, max_depth, , );

rtBuffer< rtCallableProgramId<void(MaterialParameter &mat, State &state, PerRayData_radiance &prd)> > sysBRDFPdf;
rtBuffer< rtCallableProgramId<void(MaterialParameter &mat, State &state, PerRayData_radiance &prd)> > sysBRDFSample;
rtBuffer< rtCallableProgramId<float3(MaterialParameter &mat, State &state, PerRayData_radiance &prd)> > sysBRDFEval;
rtBuffer< rtCallableProgramId<void(const LightParameter &light, const float3 &surfacePos, unsigned int &seed, LightSample &lightSample)> > sysLightSample;

rtBuffer<MaterialParameter> sysMaterialParameters;
rtDeclareVariable(int, materialId, , );
rtDeclareVariable(int, sysNumberOfLights, , );

rtBuffer<LightParameter> sysLightParameters;


RT_FUNCTION float3 DirectLight(MaterialParameter &mat, State &state)
{
	float3 L = make_float3(0.0f);

	//Pick a light to sample
	int index = optix::clamp(static_cast<int>(floorf(rnd(prd.seed) * sysNumberOfLights)), 0, sysNumberOfLights - 1);
	LightParameter light = sysLightParameters[index];
	LightSample lightSample;

	float3 surfacePos = state.fhp;

	sysLightSample[light.lightType](light, surfacePos, prd.seed, lightSample);

	if (0.0f < lightSample.pdf)
	{
		prd.bsdfDir = lightSample.direction;
		sysBRDFPdf[mat.brdf](mat, state, prd);
		float3 f = sysBRDFEval[mat.brdf](mat, state, prd);

		if (0.0f < prd.pdf && (f.x != 0.0f || f.y != 0.0f || f.z != 0.0f))
		{
			PerRayData_shadow prdShadow;
			prdShadow.inShadow = false;
			Ray shadowRay = make_Ray(surfacePos, lightSample.direction, 1, scene_epsilon, lightSample.distance - scene_epsilon);
			rtTrace(top_object, shadowRay, prdShadow);

			if (!prdShadow.inShadow)
			{
				const float misWeight = powerHeuristic(lightSample.pdf, prd.pdf);
				L = misWeight * prd.throughput * f * lightSample.emission / max(1e-3f, lightSample.pdf);
			}
		}
	}

	return L;
}


RT_PROGRAM void closest_hit()
{
	const float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	const float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	const float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

	MaterialParameter mat = sysMaterialParameters[materialId];

	if (mat.albedoID != RT_TEXTURE_ID_NULL)
	{
		const float3 texColor = make_float3(optix::rtTex2D<float4>(mat.albedoID, texcoord.x, texcoord.y));
		mat.color = make_float3(powf(texColor.x, 2.2f), powf(texColor.y, 2.2f), powf(texColor.z, 2.2f));
	}

	State state;
	state.fhp = front_hit_point;
	state.bhp = back_hit_point;
	state.normal = world_shading_normal;
	state.ffnormal = ffnormal;
	prd.wo = -ray.direction;

	// Emissive radiance
	prd.radiance += mat.emission * prd.throughput;

	// TODO: Clean up handling of specular bounces
	prd.specularBounce = mat.brdf == GLASS || mat.brdf == ROUGHDIELECTRIC ? true : false;

	// Direct light Sampling
	if (!prd.specularBounce && prd.depth < max_depth)
		prd.radiance += DirectLight(mat, state);

	// BRDF Sampling
	sysBRDFSample[mat.brdf](mat, state, prd);
	sysBRDFPdf[mat.brdf](mat, state, prd);
	float3 f = sysBRDFEval[mat.brdf](mat, state, prd);

	prd.albedo = mat.color; 
	prd.normal = ffnormal;

	if (prd.pdf > 0.0f)
	{
		prd.throughput *= f / prd.pdf;
	}
	else
	{
		prd.done = true;
	}
}


RT_PROGRAM void any_hit()
{
	prd_shadow.inShadow = true;
	rtTerminateRay();
}
