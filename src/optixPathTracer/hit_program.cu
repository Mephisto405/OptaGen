#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu_matrix_namespace.h>
#include "helpers.h"
#include "prd.h"
#include "random.h"
#include "rt_function.h"
#include "material_parameters.h"
#include "light_parameters.h"
#include "state.h"

using namespace optix;

rtDeclareVariable( float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable( float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable( float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable(float3, back_hit_point, attribute back_hit_point, );
rtDeclareVariable( float3, texcoord, attribute texcoord, );

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(int, max_depth, , );

rtBuffer< rtCallableProgramId<void(MaterialParameter &mat, State &state, PerRayData_radiance &prd)> > sysBRDFPdf;
rtBuffer< rtCallableProgramId<void(MaterialParameter &mat, State &state, PerRayData_radiance &prd)> > sysBRDFSample;
rtBuffer< rtCallableProgramId<float3(MaterialParameter &mat, State &state, PerRayData_radiance &prd)> > sysBRDFEval;
rtBuffer< rtCallableProgramId<void(LightParameter &light, PerRayData_radiance &prd, LightSample &sample)> > sysLightSample;

rtBuffer<MaterialParameter> sysMaterialParameters;
rtDeclareVariable(int, materialId, , ); 
rtDeclareVariable(int, sysNumberOfLights, , );

rtBuffer<LightParameter> sysLightParameters;

RT_FUNCTION float3 DirectLight(MaterialParameter &mat, State &state)
{
	float3 L = make_float3(0.0f);

	//Pick a light to sample
	int index = optix::clamp(static_cast<int>(floorf(rnd(prd.seed) * sysNumberOfLights)), 0, sysNumberOfLights - 1);
	LightParameter light = sysLightParameters[index];
	LightSample lightSample;

	float3 surfacePos = state.fhp;
	float3 surfaceNormal = state.ffnormal;

	sysLightSample[light.lightType](light, prd, lightSample);

	float3 lightDir = lightSample.surfacePos - surfacePos;
	float lightDist = length(lightDir);
	float lightDistSq = lightDist * lightDist;
	lightDir /= sqrtf(lightDistSq);

	if (dot(lightDir, surfaceNormal) <= 0.0f || dot(lightDir, lightSample.normal) >= 0.0f)
		return L;

	PerRayData_shadow prd_shadow;
	prd_shadow.inShadow = false;
	optix::Ray shadowRay = optix::make_Ray(surfacePos, lightDir, 1, scene_epsilon, lightDist - scene_epsilon);
	rtTrace(top_object, shadowRay, prd_shadow);

	if (!prd_shadow.inShadow)
	{
		float NdotL = dot(lightSample.normal, -lightDir);
		float lightPdf = lightDistSq / (light.area * NdotL);

		prd.bsdfDir = lightDir;

		sysBRDFPdf[mat.brdf](mat, state, prd);
		float3 f = sysBRDFEval[mat.brdf](mat, state, prd);

		L = powerHeuristic(lightPdf, prd.pdf) * prd.throughput * f * lightSample.emission / max(0.001f, lightPdf);
	}

	return L;
}

RT_PROGRAM void closest_hit()
{
	const float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	const float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	const float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

	MaterialParameter mat = sysMaterialParameters[materialId];

	if (mat.albedoID != RT_TEXTURE_ID_NULL)
	{
		const float3 texColor = make_float3(optix::rtTex2D<float4>(mat.albedoID, texcoord.x, texcoord.y));
		mat.color = make_float3(powf(texColor.x, 2.2f), powf(texColor.y, 2.2f), powf(texColor.z, 2.2f));
	}

	State state;
	state.fhp = front_hit_point;
	state.bhp = back_hit_point;
	state.normal = world_shading_normal;
	state.ffnormal = ffnormal;
	prd.wo = -ray.direction;

	prd.radiance += mat.emission * prd.throughput;

	//TODO: Clean up handling of specular bounces
	prd.specularBounce = mat.brdf == GLASS || mat.brdf == ROUGHDIELECTRIC ? true : false;

	// Direct light Sampling
	if (!prd.specularBounce && prd.depth < max_depth)
		prd.radiance += DirectLight(mat, state);

	// BRDF Sampling
	sysBRDFSample[mat.brdf](mat, state, prd);
	sysBRDFPdf[mat.brdf](mat, state, prd);
	float3 f = sysBRDFEval[mat.brdf](mat, state, prd);

	if (prd.pdf > 0.0f)
		prd.throughput *= f / prd.pdf; 
	else
		prd.done = true;
}

RT_PROGRAM void any_hit()
{
	prd_shadow.inShadow = true;
	rtTerminateRay();
}
