#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "prd.h"
#include "samplerecord.h"
#include "configs.h"
#include "rt_function.h"
#include "random.h"
#include <assert.h>
#include <stdio.h>

using namespace optix;

rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(float3, bad_color, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(float3, cutoff_color, , );
rtDeclareVariable(float, scene_radius, , );
rtBuffer<float4, 2>              output_buffer;
rtBuffer<SampleRecord[MAX_SAMPLES], 2>      mbpf_buffer; /* Multiple-bounced feature buffer */
rtBuffer<float4, 2>              accum_buffer;
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(unsigned int, frame, , );
rtDeclareVariable(unsigned int, curr_time, , );
rtDeclareVariable(int, mbpf_frames, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );


__device__ inline float4 ToneMap(const float4& c, float limit)
{
	float luminance = 0.3f*c.x + 0.6f*c.y + 0.1f*c.z;

	float4 col = c * 1.0f / (1.0f + luminance / limit);
	return make_float4(col.x, col.y, col.z, 1.0f);
}

__device__ inline float4 LinearToSrgb(const float4& c)
{
	const float kInvGamma = 1.0f / 2.2f;
	return make_float4(powf(c.x, kInvGamma), powf(c.y, kInvGamma), powf(c.z, kInvGamma), c.w);
}

__device__ inline float3 ToneMap(const float3& c, float limit)
{
	float luminance = 0.3f*c.x + 0.6f*c.y + 0.1f*c.z;

	float3 col = c * 1.0f / (1.0f + luminance / limit);
	return make_float3(col.x, col.y, col.z);
}

__device__ inline float3 LinearToSrgb(const float3& c)
{
	const float kInvGamma = 1.0f / 2.2f;
	return make_float3(powf(c.x, kInvGamma), powf(c.y, kInvGamma), powf(c.z, kInvGamma));
}

__device__ inline float3 clip(const float3& c)
{
	return make_float3(
		c.x < 0 ? 0 : c.x > 1.0 ? 1.0 : c.x,
		c.y < 0 ? 0 : c.y > 1.0 ? 1.0 : c.y,
		c.z < 0 ? 0 : c.z > 1.0 ? 1.0 : c.z
		);
}

RT_PROGRAM void pinhole_camera()
{
	/* Sub-pixel jittering */
	size_t2 screen = output_buffer.size();
	unsigned int seed = tea<16>(screen.x*launch_index.y + launch_index.x, frame + curr_time);

	float subpixel_x = frame == 0 ? 0.5f : rnd(seed);
	float subpixel_y = frame == 0 ? 0.5f : rnd(seed);
	float2 subpixel_jitter = make_float2(subpixel_x - 0.5f, subpixel_y - 0.5f);
	float2 d = (make_float2(launch_index) + subpixel_jitter) / make_float2(screen) * 2.f - 1.f;
	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x*U + d.y*V + W);


	/* Records */
	// ray records
	PerRayData_radiance prd = {};
	prd.seed = seed;
	prd.throughput = make_float3(1.0f);
	prd.throughput_diffuse = make_float3(1.0f);

	// sample records
	SampleRecord sr = {};
	sr.subpixel_x = subpixel_x;
	sr.subpixel_y = subpixel_y;
	float depth_norm = scene_radius > 0.0f ? 1.0f / (10.0f * scene_radius) : 1.0f;


	/* Main rendering loop */
	float3 result = make_float3(0.0f);
	for (;;) {
		optix::Ray ray(ray_origin, ray_direction, 0 /*ray type*/, scene_epsilon);
		prd.wo = -ray.direction;
		rtTrace(top_object, ray, prd);


		/* post-processing */
		// at the first geometric bounce
		if (prd.depth == 0)
		{
			sr.albedo_at_first = prd.albedo;
			sr.normal_at_first = prd.normal;
			sr.depth_at_first = prd.hasHit ? prd.ray_dist * depth_norm : -0.1f;
			sr.visibility = prd.hasHit ? (!prd.inShadow ? 1.0f : 0.0f) : 0.0f;
			sr.hasHit = prd.hasHit ? 1.0f : 0.0f;
		}
		
		// TODO(iycho): dirty code and not work properly
		if (prd.depth == 1 && !prd.hasHit && dot(prd.light_intensity, prd.light_intensity) != 0)
		{
			// the object is visible if the ray hit a non-black light at the second bounce
			sr.visibility = true;
		}

		// either at the first non specular bounce 
		// or no specular bounce until the end of light transport
		if (prd.is_first_non_specular || (sr.depth == 0.0f && !prd.hasHit))
		{
			sr.albedo = prd.albedo;
			sr.normal = prd.normal;
			sr.depth = prd.hasHit ? prd.ray_dist * depth_norm : -0.1f;
		}

		if (!prd.hasHit)
		{
			sr.light_intensity = prd.light_intensity;
		}
		else
		{
			if (prd.depth == 0)
			{
				sr.path_weight = 1.0f;
			}
			sr.path_weight *= prd.pdf;
			sr.radiance_wo_weight *= prd.thpt_at_vtx;

			// record sample data
			sr.throughputs[prd.depth] = prd.thpt_at_vtx;
			sr.bounce_types[prd.depth] = (float)prd.bounce_type;
			sr.roughnesses[prd.depth] = prd.roughness;
		}

		/* exit if light cannot transfer further */
		if (prd.done || prd.depth >= MAX_DEPTH) // >= max_depth
			break;

		prd.depth++;

		// update ray data for the next path segment
		ray_origin = prd.origin;
		ray_direction = prd.bsdfDir;
	}

	sr.radiance = prd.radiance;
	sr.radiance_diffuse = prd.radiance_diffuse;
	result = prd.radiance;

	float4 acc_val = accum_buffer[launch_index];
	if (frame > 0) {
		acc_val = lerp(acc_val, make_float4(result, 0.f), 1.0f / static_cast<float>(frame + 1));
	}
	else {
		acc_val = make_float4(result, 0.f);
	}

	//float4 val = LinearToSrgb(ToneMap(acc_val, 1.5));
	//float4 val = LinearToSrgb(acc_val);

	output_buffer[launch_index] = acc_val; // uint
	accum_buffer[launch_index] = acc_val;
	if (frame < mbpf_frames)
		mbpf_buffer[launch_index][frame] = sr;
}

RT_PROGRAM void exception()
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf("Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y);
	output_buffer[launch_index] = make_float4(bad_color);
}