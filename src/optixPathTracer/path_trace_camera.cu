#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "prd.h"
#include "rt_function.h"
#include "random.h"

using namespace optix;


rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(float3,        cutoff_color, , );
rtDeclareVariable(int,           max_depth, , );
rtBuffer<uchar4, 2>              output_buffer;
rtBuffer<float3, 2>              normal_buffer;
rtBuffer<float4, 2>              accum_buffer;
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(unsigned int,  frame, , );
rtDeclareVariable(uint2,         launch_index, rtLaunchIndex, );

__device__ inline float4 ToneMap(const float4& c, float limit)
{
	float luminance = 0.3f*c.x + 0.6f*c.y + 0.1f*c.z;

	float4 col = c * 1.0f / (1.0f + luminance / limit);
	return make_float4(col.x, col.y, col.z, 1.0f);
}

__device__ inline float4 LinearToSrgb(const float4& c)
{
	const float kInvGamma = 1.0f / 2.2f;
	return make_float4(powf(c.x, kInvGamma), powf(c.y, kInvGamma), powf(c.z, kInvGamma), c.w);
}


RT_PROGRAM void pinhole_camera()
{

	size_t2 screen = output_buffer.size();
	unsigned int seed = tea<16>(screen.x*launch_index.y+launch_index.x, frame);

	// Subpixel jitter: send the ray through a different position inside the pixel each time,
	// to provide antialiasing.
	float2 subpixel_jitter = frame == 0 ? make_float2( 0.0f ) : make_float2(rnd( seed ) - 0.5f, rnd( seed ) - 0.5f);

	float2 d = (make_float2(launch_index) + subpixel_jitter) / make_float2(screen) * 2.f - 1.f;
	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x*U + d.y*V + W);

	PerRayData_radiance prd;
	prd.depth = 0;
	prd.seed = seed;
	prd.done = false;
	prd.pdf = 0.0f;
	prd.specularBounce = false;

	// These represent the current shading state and will be set by the closest-hit or miss program

	// attenuation (<= 1) from surface interaction.
	prd.throughput = make_float3( 1.0f );

	// light from a light source or miss program
	prd.radiance = make_float3( 0.0f );
	prd.normal = make_float3(0.0f);

	// next ray to be traced
	prd.origin = make_float3( 0.0f );
	prd.bsdfDir = make_float3( 0.0f );

	float3 result = make_float3( 0.0f );

	// Main render loop. This is not recursive, and for high ray depths
	// will generally perform better than tracing radiance rays recursively
	// in closest hit programs.
	for(;;) {
		optix::Ray ray(ray_origin, ray_direction, /*ray type*/ 0, scene_epsilon );
		prd.wo = -ray.direction;
		rtTrace(top_object, ray, prd);

		if (prd.depth == 0 && frame == 0)
			normal_buffer[launch_index] = (prd.normal.x == 0.f && prd.normal.y == 0.f && prd.normal.z == 0.f) ? 
										  prd.normal : 
										  0.5f * normalize(prd.normal) + make_float3(0.5f); // normalize(prd.normal)

		if (prd.done || prd.depth >= max_depth)
			break;

		prd.depth++;

		// Update ray data for the next path segment
		ray_origin = prd.origin;
		ray_direction = prd.bsdfDir;
	}

	result = prd.radiance;

	float4 acc_val = accum_buffer[launch_index];
	if( frame > 0 ) {
		acc_val = lerp(acc_val, make_float4(result, 0.f), 1.0f / static_cast<float>(frame + 1));
	} else {
		acc_val = make_float4(result, 0.f);
	}

	float4 val = LinearToSrgb(ToneMap(acc_val, 1.5));
	//float4 val = LinearToSrgb(acc_val);

	output_buffer[launch_index] = make_color(make_float3(val));
	accum_buffer[launch_index] = acc_val;
}

RT_PROGRAM void exception()
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
	output_buffer[launch_index] = make_color( bad_color );
}




