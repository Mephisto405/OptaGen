#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "prd.h"
#include "path.h"
#include "rt_function.h"
#include "random.h"
#include <assert.h>
#include <stdio.h>

using namespace optix;


rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(float3, bad_color, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(float3, cutoff_color, , );
rtDeclareVariable(int, max_depth, , );
rtBuffer<float4, 2>              output_buffer;
rtBuffer<PathFeature[4], 2>      mbpf_buffer; /* Multiple-bounced feature buffer */
rtBuffer<float4, 2>              accum_buffer;
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(unsigned int, frame, , );
rtDeclareVariable(unsigned int, curr_time, , );
rtDeclareVariable(int, mbpf_frames, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );


__device__ inline float4 ToneMap(const float4& c, float limit)
{
	float luminance = 0.3f*c.x + 0.6f*c.y + 0.1f*c.z;

	float4 col = c * 1.0f / (1.0f + luminance / limit);
	return make_float4(col.x, col.y, col.z, 1.0f);
}

__device__ inline float4 LinearToSrgb(const float4& c)
{
	const float kInvGamma = 1.0f / 2.2f;
	return make_float4(powf(c.x, kInvGamma), powf(c.y, kInvGamma), powf(c.z, kInvGamma), c.w);
}

__device__ inline float3 ToneMap(const float3& c, float limit)
{
	float luminance = 0.3f*c.x + 0.6f*c.y + 0.1f*c.z;

	float3 col = c * 1.0f / (1.0f + luminance / limit);
	return make_float3(col.x, col.y, col.z);
}

__device__ inline float3 LinearToSrgb(const float3& c)
{
	const float kInvGamma = 1.0f / 2.2f;
	return make_float3(powf(c.x, kInvGamma), powf(c.y, kInvGamma), powf(c.z, kInvGamma));
}

__device__ inline float3 clip(const float3& c)
{
	return make_float3(
		c.x < 0 ? 0 : c.x > 1.0 ? 1.0 : c.x,
		c.y < 0 ? 0 : c.y > 1.0 ? 1.0 : c.y,
		c.z < 0 ? 0 : c.z > 1.0 ? 1.0 : c.z
		);
}

RT_PROGRAM void pinhole_camera()
{
	size_t2 screen = output_buffer.size();
	unsigned int seed = tea<16>(screen.x*launch_index.y + launch_index.x, frame + curr_time);

	// Subpixel jitter: send the ray through a different position inside the pixel each time,
	// to provide antialiasing.
	float2 subpixel_jitter = frame == 0 ? make_float2(0.0f) : make_float2(rnd(seed) - 0.5f, rnd(seed) - 0.5f);

	float2 d = (make_float2(launch_index) + subpixel_jitter) / make_float2(screen) * 2.f - 1.f;
	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x*U + d.y*V + W);

	PerRayData_radiance prd;
	prd.depth = 0;
	prd.seed = seed;
	prd.done = false;
	prd.pdf = 0.0f;
	prd.specularBounce = false;
	prd.thpt_at_vtx = make_float3(0.0f);
	prd.tag = DIFF;
	prd.roughness = 0.0f;


	// These represent the current shading state and will be set by the closest-hit or miss program

	// attenuation (<= 1) from surface interaction.
	prd.throughput = make_float3(1.0f);

	// light from a light source or miss program
	prd.radiance = make_float3(0.0f);

	// next ray to be traced
	prd.origin = make_float3(0.0f);
	prd.bsdfDir = make_float3(0.0f);

	float3 result = make_float3(0.0f);

	PathFeature pf{
		{ optix::make_float3(0.f) }, { DIFF }, { 0.0f }, // multi-bounce features
		make_float3(0.f), make_float3(0.f), make_float3(0.f), // first-bounce features
		1.0f, // MC probability
	};

	// Main render loop. This is not recursive, and for high ray depths
	// will generally perform better than tracing radiance rays recursively
	// in closest hit programs.
	for (;;) {
		optix::Ray ray(ray_origin, ray_direction, /*ray type*/ 0, scene_epsilon);
		prd.wo = -ray.direction;
		rtTrace(top_object, ray, prd);

		if (prd.depth == 0)
		{
			pf.albedo = clip(prd.albedo);
			pf.normal = (prd.normal.x == 0.f && prd.normal.y == 0.f && prd.normal.z == 0.f) ?
				prd.normal :
				0.5f * normalize(prd.normal) + 0.5f;
		}

		if (prd.done)
			break;

		/* Path features */
		pf.prob *= prd.pdf;
		if (prd.depth < 6)
		{
			pf.throughput[prd.depth] = prd.thpt_at_vtx;
			pf.tag[prd.depth] = (float)prd.tag;
			pf.roughness[prd.depth] = prd.roughness;
		}
		else
		{
			pf.throughput[5] *= prd.thpt_at_vtx;
		}

		if (prd.done || prd.depth >= max_depth)
			break;

		prd.depth++;

		// Update ray data for the next path segment
		ray_origin = prd.origin;
		ray_direction = prd.bsdfDir;
	}

	pf.radiance = prd.radiance;
	result = prd.radiance;

	float4 acc_val = accum_buffer[launch_index];
	if (frame > 0) {
		acc_val = lerp(acc_val, make_float4(result, 0.f), 1.0f / static_cast<float>(frame + 1));
	}
	else {
		acc_val = make_float4(result, 0.f);
	}

	//float4 val = LinearToSrgb(ToneMap(acc_val, 1.5));
	//float4 val = LinearToSrgb(acc_val);

	output_buffer[launch_index] = acc_val; // uint
	accum_buffer[launch_index] = acc_val;
	if (frame < mbpf_frames)
		mbpf_buffer[launch_index][frame] = pf;
}

RT_PROGRAM void exception()
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf("Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y);
	output_buffer[launch_index] = make_float4(bad_color);
}