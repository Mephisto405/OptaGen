#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived 
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "prd.h"
#include "helpers.h"
#include "light_parameters.h"
#include <assert.h>

using namespace optix;

rtDeclareVariable(float3, background_light, , ); // horizon color
rtDeclareVariable(float3, background_dark, , );  // zenith color
rtDeclareVariable(float3, up, , );               // global up vector
rtDeclareVariable(int, option, , );				 // 1 if the envmap is given, 0 otherwise

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );

rtBuffer<LightParameter> sysLightParameters;


RT_PROGRAM void miss()
{
	if (option == 0)
	{
		prd.done = true;
	}
	else
	{		
		const LightParameter light = sysLightParameters[0];

		float3 dir = normalize(ray.direction); // might be unnecessary
		float theta = acosf(-dir.y); // theta == 0.0f is south pole, theta == M_PIf is north pole
		float phi = (dir.x == 0.0f && dir.z == 0.0f) ? 0.0f : atan2f(dir.x, -dir.z); // Starting on positive z-axis going around clockwise (to negative x-axis)
		float u = (M_PI + phi) * (0.5f * M_1_PIf) /* + 0.5f */;
		float v = theta * M_1_PIf;

		const float3 emission = make_float3(optix::rtTex2D<float4>(light.idEnvironmentTexture, u, v)); // env map support

		float misWeight = 1.0f;
		if (!prd.specularBounce && prd.depth != 0)
		{
			//assert(sysLightParameters[0].lightType != LightType::ENVMAP);
			const float pdfLight = 0.3333333333f * (emission.x + emission.y + emission.z) / light.environmentIntegral;
			misWeight = powerHeuristic(prd.pdf, pdfLight);
		}

		prd.radiance = misWeight * prd.throughput * emission;
		prd.done = true;
	}
}