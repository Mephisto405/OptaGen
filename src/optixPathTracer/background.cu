#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived 
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "prd.h"

using namespace optix;

rtDeclareVariable(float3, background_light, , ); // horizon color
rtDeclareVariable(float3, background_dark, , );  // zenith color
rtDeclareVariable(float3, up, , );               // global up vector
rtDeclareVariable(int, option, , );				 // 1 if the envmap is given, 0 otherwise

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtTextureSampler<float4, 2> envmap;

// -----------------------------------------------------------------------------

/*RT_PROGRAM void miss()
{
  const float t = max(dot(ray.direction, up), 0.0f);
  const float3 result = lerp(background_light, background_dark, t);

  prd_radiance.radiance = result;
  prd_radiance.done = true;
}

inline __device__ float3 missColor(const optix::Ray &ray)
{
	const float3 unit_direction = normalize(ray.direction);
	const float t = 0.5f * (unit_direction.y + 1.0f);
	const float3 c = (1.0f - t) * make_float3(1.0f, 1.0f, 1.0f) + t * make_float3(0.5f, 0.7f, 1.0f);
	return c;
}*/

RT_PROGRAM void miss()
{
	//const float t = max(dot(ray.direction, up), 0.0f);
	//const float3 result = lerp(background_light, background_dark, t);

	//prd_radiance.radiance = 0.01 * result;
	//prd_radiance.done = true;

	if (option == 0)
	{
		prd_radiance.done = true;
	}
	else
	{
		float theta = atan2f(ray.direction.x, ray.direction.z);
		float phi = M_PIf * 0.5f - acosf(ray.direction.y);
		float u = (theta + M_PIf) * (0.5f * M_1_PIf);
		float v = 0.5f * (1.0f + sin(phi));
		prd_radiance.radiance = prd_radiance.throughput * make_float3(tex2D(envmap, u, v)); // env map support
		prd_radiance.done = true;
	}
}