#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "prd.h"
#include "helpers.h"
#include "light_parameters.h"
#include <assert.h>

using namespace optix;

rtDeclareVariable(int, sysNumberOfLights, , );

rtDeclareVariable(float3, background_light, , ); // horizon color
rtDeclareVariable(float3, background_dark, , );  // zenith color
rtDeclareVariable(float3, up, , );               // global up vector
rtDeclareVariable(int, option, , );				 // 1 if the envmap is given, 0 otherwise

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );

rtBuffer<LightParameter> sysLightParameters;


__device__ inline float3 ToneMap(const float3& c, float limit)
{
	float luminance = 0.3f*c.x + 0.6f*c.y + 0.1f*c.z;

	float3 col = c * 1.0f / (1.0f + luminance / limit);
	return make_float3(col.x, col.y, col.z);
}

__device__ inline float3 LinearToSrgb(const float3& c)
{
	const float kInvGamma = 1.0f / 2.2f;
	return make_float3(powf(c.x, kInvGamma), powf(c.y, kInvGamma), powf(c.z, kInvGamma));
}


RT_PROGRAM void miss()
{
	if (option != 0)
	{
		const LightParameter light = sysLightParameters[sysNumberOfLights - 1];
		assert(light.lightType == ENVMAP);

		float3 dir = normalize(ray.direction); // might be unnecessary
		float theta = acosf(-dir.y); // theta == 0.0f is south pole, theta == M_PIf is north pole
		float phi = (dir.x == 0.0f && dir.z == 0.0f) ? 0.0f : atan2f(dir.x, -dir.z); // Starting on positive z-axis going around clockwise (to negative x-axis)
		float u = (M_PI + phi) * (0.5f * M_1_PIf) /* + 0.5f */;
		float v = theta * M_1_PIf;

		const float3 emission = make_float3(optix::rtTex2D<float4>(light.idEnvironmentTexture, u, v)); // env map support

		float misWeight = 1.0f;
		if (!prd.specularBounce && prd.depth != 0)
		{
			//assert(sysLightParameters[0].lightType != LightType::ENVMAP);
			const float pdfLight = 0.3333333333f * (emission.x + emission.y + emission.z) / light.environmentIntegral;
			misWeight = powerHeuristic(prd.pdf, pdfLight);
		}

		prd.light_intensity = emission;
		prd.radiance += misWeight * emission * prd.throughput;
	}

	prd.albedo = make_float3(0.f);
	prd.normal = make_float3(0.f);
	prd.hasHit = false;

	prd.done = true;
}