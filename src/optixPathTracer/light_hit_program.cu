#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu_matrix_namespace.h>
#include "helpers.h"
#include "prd.h"
#include "random.h"
#include "rt_function.h"
#include "light_parameters.h"
#include "state.h"

using namespace optix;

rtDeclareVariable( float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable( float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable( float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable(float3, back_hit_point, attribute back_hit_point, );

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, hit_dist, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(float, scene_epsilon, , );

rtBuffer<LightParameter> sysLightParameters;
rtDeclareVariable(int, lightMaterialId, , );

RT_PROGRAM void closest_hit()
{
	const float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	const float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
	const float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

	LightParameter light = sysLightParameters[lightMaterialId];
	float cosTheta = dot(-ray.direction, light.normal);

	if ((light.lightType == QUAD && cosTheta > 0.0f) || light.lightType == SPHERE)
	{
		if(prd.depth == 0 || prd.specularBounce)
			prd.radiance += light.emission * prd.throughput;
		else
		{
			float lightPdf = (hit_dist * hit_dist) / (light.area * clamp(cosTheta, 1.e-3f, 1.0f));
			prd.radiance += powerHeuristic(prd.pdf, lightPdf) * prd.throughput * light.emission;
		}
	}

	prd.done = true;
}